
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <memory>
#include <cstdint>
#include <filesystem>
#include <string>

namespace solution {
    #define CUDA_ERROR_CHECK(ans) { cudaAssert((ans), __FILE__, __LINE__); } 
    inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true) {
        if (code != hipSuccess) {
            fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
            if (abort) exit(code);
        }
    }

    __global__ void convolution2D(const float *input_image, float *output_result, const float *conv_kernel, const int N_Rows, const int N_Cols) {
        int bx = blockIdx.x;
        int by = blockIdx.y;
                
        int tx = threadIdx.x;
        int thread_id = bx * blockDim.x + tx;
        int s = N_Rows * N_Cols;
        if(thread_id < s) {
            float sum = 0.0;
            int i = thread_id / N_Cols;
            int j = thread_id % N_Cols;
            for(int d_row = -1; d_row <= 1; d_row++) {
                for(int d_col = -1; d_col <= 1; d_col++) {
                    int new_row = i + d_row, new_col = j + d_col;
                    if(new_row >= 0 && new_row < N_Rows && new_col >= 0 && new_col < N_Cols) {
                        sum += conv_kernel[(d_row + 1) * 3 + (d_col + 1)] * input_image[new_row * N_Cols + new_col];
                    }
                }
            }
            output_result[thread_id] = sum;
        }
    }

    std::string compute(const std::string &bitmap_path, const float kernel[3][3], const std::int32_t N_Rows, const std::int32_t N_Cols) {
        std::string solution_path = std::filesystem::temp_directory_path() / "student_solution.bmp";
        std::ofstream sol_fs(solution_path, std::ios::binary);
        std::ifstream bitmap_fs(bitmap_path, std::ios::binary);
        int s = N_Rows * N_Cols;
        const auto image = std::make_unique<float[]>(s);
        bitmap_fs.read(reinterpret_cast<char*>(image.get()), sizeof(float) * s);

        float *device_image, *device_result, *device_kernel;
        CUDA_ERROR_CHECK(hipMalloc((void **)&device_image, sizeof(float) * s));
        CUDA_ERROR_CHECK(hipMalloc((void **)&device_result, sizeof(float) * s));
        CUDA_ERROR_CHECK(hipMalloc((void **)&device_kernel, sizeof(float) * 3 * 3));

        CUDA_ERROR_CHECK(hipMemcpy(device_image, image.get(), sizeof(float) * s, hipMemcpyHostToDevice));
        CUDA_ERROR_CHECK(hipMemcpy(device_kernel, kernel, sizeof(float) * 3 * 3, hipMemcpyHostToDevice));

        int size_block = 512;
        int Num_Blocks = (N_Rows * N_Cols + size_block - 1) / size_block;

        convolution2D<<<Num_Blocks, size_block>>>(device_image, device_result, device_kernel, N_Rows, N_Cols);

        float *result = new float[s];
        CUDA_ERROR_CHECK(hipMemcpy(result, device_result, sizeof(float) * s, hipMemcpyDeviceToHost));

        sol_fs.write(reinterpret_cast<char*>(result), sizeof(float) * s);

        hipFree(device_image);
        hipFree(device_result);
        hipFree(device_kernel);

        delete[] result;

        return solution_path;
    }
};
